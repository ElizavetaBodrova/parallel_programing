#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <random>
#include <chrono>

__device__ bool isPrime(const int number) {
    int divider = 2;
    bool flag = true;
    while (divider < number - 1 && flag) {
        flag = number % divider != 0;
        divider++;
    }
    return flag;
}


bool isPrimeNonParallel(const int number) {
    int divider = 2;
    bool flag = true;
    while (divider < number - 1 && flag) {
        flag = number % divider != 0;
        divider++;
    }
    return flag;
}

int findCountNonParallel(int *arr, int N) {
    int localCount = 0;
    for (int i = 0; i < N * N; i++) {
        if (isPrimeNonParallel(arr[i])) {
            localCount++;
        }
    }
    return localCount;
}

//каждый поток
__global__ void _primeCount(int *matrix, const int N, int *result) {
    //рассчитывание координат

    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;

    while (globalIdx < N) {
        if (isPrime(matrix[globalIdx])) {
            atomicAdd(&(result[0]), 1);
            matrix[globalIdx] = 0;
        }
        globalIdx += blockDim.x * gridDim.x;
        __syncthreads();

    }

}

void initMatrix(int *matrix, const int N) {
        srand(time(NULL));
        for (int i = 0; i < N * N; ++i) {
            matrix[i] = 1 + rand() % 10;
          //  std::cout << matrix[i] << " ";
        }
      //  std::cout << std::endl;
    }

int main() {
    //const int N = 1 << 10; // 1024
    const int N = 8; // 24
    std::cout << "N = " << N << std::endl;
    size_t bytes = (N * N) * sizeof(N);
    int *matrix;
    hipMallocManaged(&matrix, bytes);
    initMatrix(matrix, N);

    std::cout << "NonParallel " << findCountNonParallel(matrix, N * N) % (N * N) << std::endl;

// Copy data to device
    int globalResult = 0;
    int* d_test_data, * global;
    hipMalloc(&global, sizeof(int));
    hipMalloc(&d_test_data, N * sizeof(int));
    hipMemcpy(global, &globalResult, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_test_data, matrix, N*N * sizeof(int), hipMemcpyHostToDevice);

// Launch kernel
    _primeCount << <10, 1024 >> > (matrix,N*N, global);

// Copy results back to device
    hipDeviceSynchronize();
    hipMemcpy(matrix, d_test_data, N*N * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&globalResult, global, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_test_data);
    hipFree(global);

    std::cout << "Work is done! " << globalResult << std::endl;
    for (int i = 0; i <N*N; ++i) {
        std::cout << matrix[i] << " ";
    }
    std::cout << std::endl;
    system("pause");
    return 0;
}
